
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdbool.h>
#include <sys/time.h>

typedef int32_t Int;


#define NUM_BYTES(n) ((n) * (sizeof(Int)))

__global__ void compressedRow(Int* matrix, Int* rowIndex, Int* nums, Int* cols, Int* rows, Int width) {
	Int* row = ((matrix + (blockIdx.x * width)));
	Int offset = *(rowIndex + blockIdx.x);

	Int i = 0;
	for (; i < width; i++) {
		if (*(row + i) != 0) {
			*(nums + offset) = *(row + i);
			*(cols + offset) = i;
			*(rows + offset) = blockIdx.x;
			offset++;
		}
	}
}

__global__ void decompressRow(Int* matrix, Int* nums, Int* cols, Int* rows, Int count, Int width) {
	Int* row = ((matrix + (blockIdx.x * width)));
	Int i;

    *(row + threadIdx.x) = 0;
	for (i = 0; i < count; i++) {
		if (*(rows + i) == blockIdx.x) {
			if (*(cols + i) == threadIdx.x) {
				*(row + threadIdx.x) = *(nums + i);
			}
		}
	}

}

bool fileExists (char* name) {
   FILE* tmp   = fopen (name, "rb");
   bool exists = (tmp != NULL);
   if (tmp != NULL) fclose (tmp);
   return exists;
}

void compress(char* infile) {
	
	FILE* time_results = fopen("sectionResultsCompress.csv", "a+");
	struct timeval stop, start;

	gettimeofday(&start, NULL); //BEGIN READ-IN SECTION
	Int height, width;
	Int count = 0, rowCount = 0;
	Int i, j;
	Int temp;

	FILE* in = fopen(infile, "rb");

	Int result = fread((void*)&height, NUM_BYTES(1), 1, in);
	if (result != 1) {
		fprintf(stderr, "Reading Error\n");
		exit(1);
	}
	result = fread((void*)&width, NUM_BYTES(1), 1, in);
	if (result != 1) {
		fprintf(stderr, "Reading Error\n");
		exit(1);
	}
	Int rowIndex[height];
	Int* matrix = (Int*)malloc(NUM_BYTES(height * width));
	for (i = 0; i < height; i++) {

		if (i == 0)
			rowIndex[i] = 0;
		else 
			rowIndex[i] = rowIndex[i-1] + rowCount;
		rowCount = 0;

		for (j = 0; j < width; j++) {
			result = fread((void*)&temp, NUM_BYTES(1), 1, in);
			*(matrix + (i * width) + j) = temp;
			if (result != 1) {
				fprintf(stderr, "Reading Error\n");
				exit(1);
			}
			if (*(matrix + (i * width) + j) != 0) {
				count++;
				rowCount++;
			}
		}
	}
	fclose(in);
	
	gettimeofday(&stop, NULL); //END READ-IN SECTION
	long timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li,", timeMillies);

	gettimeofday(&start, NULL); //BEGIN CPU-TO-GPU SECTION
	Int* cRowIndex;
	Int* cnums;
	Int* nums;
	Int* cRow; 
	Int* row;
	Int* cCol; 
	Int* col;
	Int* cMatrix;
	//size_t pitch;

	//cudaError_t cerr;
	hipMalloc((void**)&cRowIndex, NUM_BYTES(height));
	hipMalloc((void**)&cnums, NUM_BYTES(count));
	hipMalloc((void**)&cRow, NUM_BYTES(count));
	hipMalloc((void**)&cCol, NUM_BYTES(count));
	//if (cerr != cudaSuccess)
	//	fprintf(stderr, "Error with cCol malloc: %s", cudaGetErrorString(cerr));
	//cudaMallocPitch((void**)&cMatrix, &pitch, (size_t)NUM_BYTES(width), (size_t)NUM_BYTES(height));
	hipMalloc((void**)&cMatrix, NUM_BYTES(width * height));
	//if (cerr != cudaSuccess)
	//	fprintf(stderr, "Error with cCol malloc: %s", cudaGetErrorString(cerr));

	hipMemcpy(cRowIndex, rowIndex, NUM_BYTES(height), hipMemcpyHostToDevice);
	//cudaMemcpy2D((void*)cMatrix, pitch, matrix, NUM_BYTES(width), NUM_BYTES(width), NUM_BYTES(height), cudaMemcpyHostToDevice);
	hipMemcpy(cMatrix, matrix, NUM_BYTES(width * height), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	gettimeofday(&stop, NULL); //END CPU-TO-GPU SECTION
	timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li,", timeMillies);\

	gettimeofday(&start, NULL); //BEGIN GPU-COMPRESSION SECTION
	compressedRow<<<height, 1>>>(cMatrix, cRowIndex, cnums, cCol, cRow, width);
	hipDeviceSynchronize();
	gettimeofday(&stop, NULL); //END GPU-COMPRESSION SECTION
	timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li,", timeMillies);

	gettimeofday(&start, NULL); //BEGIN GPU-TO-CPU SECTION
	nums = (Int*)malloc(NUM_BYTES(count));
	row = (Int*)malloc(NUM_BYTES(count));
	col = (Int*)malloc(NUM_BYTES(count));

	hipMemcpy(nums, cnums, NUM_BYTES(count), hipMemcpyDeviceToHost);
	hipMemcpy(col, cCol, NUM_BYTES(count), hipMemcpyDeviceToHost);
	hipMemcpy(row, cRow, NUM_BYTES(count), hipMemcpyDeviceToHost);

	hipFree(cRowIndex);
	hipFree(cnums);
	hipFree(cRow);
	hipFree(cCol);	
	hipFree(cMatrix);
	hipDeviceSynchronize();

	gettimeofday(&stop, NULL); //END GPU-TO-CPU SECTION
	timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li,", timeMillies);

	gettimeofday(&start, NULL); //BEGIN WRITE-OUT SECTION
	char name[64];
	sprintf(name, "%s.crs", infile);
	FILE* file = fopen(name, "ab+");
	fwrite((void*)&height, NUM_BYTES(1), 1, file);
	fwrite((void*)&width, NUM_BYTES(1), 1, file);
	fwrite((void*)&count, NUM_BYTES(1), 1, file);

	fwrite((void*)&nums[0], NUM_BYTES(1), count, file);
	fwrite((void*)&col[0], NUM_BYTES(1), count, file);
	fwrite((void*)&row[0], NUM_BYTES(1), count, file);

	fclose(file);

	free(nums);
	free(row);
	free(col);
	free(matrix);
	gettimeofday(&stop, NULL); //END WRITE-OUT-SECTION
	timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li\n", timeMillies);
	fclose(time_results);	
}

void uncompress(char* infile) {

	FILE* time_results = fopen("sectionResultsUncompress.csv", "a+");

	struct timeval start, stop;

	gettimeofday(&start, NULL); //BEGIN READ-IN SECTION
	Int height, width;
	Int count = 0;
	Int i;
	Int temp;

	FILE* in = fopen(infile, "r");

	Int result = fread((void*)&height, NUM_BYTES(1), 1, in);
	if (result != 1) {
		fprintf(stderr, "Reading Error\n");
		exit(1);
	}
	result = fread((void*)&width, NUM_BYTES(1), 1, in);
	if (result != 1) {
		fprintf(stderr, "Reading Error\n");
		exit(1);
	}

	result = fread((void*)&count, NUM_BYTES(1), 1, in);
	if (result != 1) {
		fprintf(stderr, "Reading Error\n");
		exit(1);
	}
	Int nums[count];
	Int rows[count];
	Int cols[count];

	for (i = 0; i < count; i++) {
		result = fread((void*)&temp, NUM_BYTES(1), 1, in);
		nums[i] = temp;
		if (result != 1) {
			fprintf(stderr, "Reading Error\n");
			exit(1);
		}
	}
	for (i = 0; i < count; i++) {
		result = fread((void*)&temp, NUM_BYTES(1), 1, in);
		cols[i] = temp;
		if (result != 1) {
			fprintf(stderr, "Reading Error\n");
			exit(1);
		}
	}
	for (i = 0; i < count; i++) {
		result = fread((void*)&temp, NUM_BYTES(1), 1, in);
		rows[i] = temp;
		if (result != 1) {
			fprintf(stderr, "Reading Error\n");
			exit(1);
		}
	}
	fclose(in);

	gettimeofday(&stop, NULL); //END READ-IN SECTION
	long timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li,", timeMillies);

	gettimeofday(&start, NULL); //BEGIN CPU-TO-GPU SECTION
	Int* cNums;
	Int* cRows;
	Int* cCols;
	Int* matrix = (Int*)malloc(NUM_BYTES(height * width));
	Int* cMatrix;
	//size_t pitch;

	//cudaError_t cerr;
	hipMalloc((void**)&cNums, NUM_BYTES(count));
	hipMalloc((void**)&cRows, NUM_BYTES(count));
	hipMalloc((void**)&cCols, NUM_BYTES(count));
	//cudaMallocPitch((void**)&cMatrix, &pitch, NUM_BYTES(width), NUM_BYTES(height));
	hipMalloc((void**)&cMatrix, NUM_BYTES(width * height));

	hipMemcpy(cNums, nums, NUM_BYTES(count), hipMemcpyHostToDevice);
	hipMemcpy(cCols, cols, NUM_BYTES(count), hipMemcpyHostToDevice);
	hipMemcpy(cRows, rows, NUM_BYTES(count), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	gettimeofday(&stop, NULL); //END CPU-TO-GPU SECTION
	timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li,", timeMillies);

	gettimeofday(&start, NULL); //BEGIN GPU-COMPRESSION SECTION

	decompressRow<<<height, width>>>(cMatrix, cNums, cCols, cRows, count, width);
	hipDeviceSynchronize();

	gettimeofday(&stop, NULL); //END GPU-COMPRESSION SECTION
	timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li,", timeMillies);

	gettimeofday(&start, NULL); //BEGIN GPU-TO-CPU SECTION

	//cudaMemcpy2D((void*)matrix, NUM_BYTES(width), cMatrix, pitch, NUM_BYTES(width), NUM_BYTES(height), cudaMemcpyDeviceToHost);
	hipMemcpy((void*)matrix, cMatrix, NUM_BYTES(width * height), hipMemcpyDeviceToHost);
	//cudaMemcpy(row, cRow, NUM_BYTES(count), cudaMemcpyDeviceToHost);
	hipFree(cNums);
	hipFree(cRows);
	//if (cerr != cudaSuccess)
	//	fprintf(stderr, "Error with cRows free: %s\n", cudaGetErrorString(cerr));
	hipFree(cCols);
	//if (cerr != cudaSuccess)
	//	fprintf(stderr, "Error with cCols free: %s\n", cudaGetErrorString(cerr));
	hipFree(cMatrix);

	hipDeviceSynchronize();

	gettimeofday(&stop, NULL); //END GPU-TO-CPU SECTION
	timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li,", timeMillies);

	gettimeofday(&start, NULL); //BEGIN WRITE-OUT SECTION

	char name[64];
	sprintf(name, "%s.out", infile);
	FILE* newfile = fopen(name, "ab+");
	fwrite((void*)&height, NUM_BYTES(1), 1, newfile);
	fwrite((void*)&width, NUM_BYTES(1), 1, newfile);

	//this section could be done without the nested fors, but I already got test results
	//for the end-to-end time tests, and I have to keep it this way for research integrity
	Int j;
	for (i = 0; i < height; i++) {
		for (j = 0; j < width; j++) {
			fwrite((void*)(matrix + (i * width) + j), NUM_BYTES(1), 1, newfile);
		}
	}
	fclose(newfile);
	//free(matrix);

	gettimeofday(&stop, NULL); //END WRITE-OUT-SECTION
	timeMillies = ((stop.tv_usec) + (stop.tv_sec * 1000000)) - ((start.tv_usec) + (start.tv_sec * 1000000));
	fprintf(time_results, "%li\n", timeMillies);
	fclose(time_results);

}


int main(int argc, char* argv[]) {

	if (argc != 3 || (strcmp(argv[1], "-c") != 0 && strcmp(argv[1], "-u") != 0)) {
		fprintf(stderr, "Usage:\n%s -c filename ....... to compress\n%s -u filename ....... to uncompress\n", argv[0], argv[0]);
		exit(1);
	}
	else if (!fileExists(argv[2])) {
		fprintf(stderr, "File %s does not exist.\n", argv[2]);
		exit(1);
	}

	if (strcmp(argv[1], "-c") == 0) {
		compress(argv[2]);
	}
	else {
		uncompress(argv[2]);
	}


	exit(0);
}

