
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

int main()
{
    hipError_t cerr;
    cerr = hipProfilerStart();
    if (cerr != hipSuccess) {
        fprintf(stdout, "Error with cudeProfilerStart with error %s\n", hipGetErrorString(cerr));
    }
    const unsigned int N = 1048576;
    const unsigned int bytes = N * sizeof(int);
    int *h_a = (int*)malloc(bytes);
    int *d_a;
    cerr = hipMalloc((int**)&d_a, bytes);
    if (cerr != hipSuccess) {
        fprintf(stderr, "Error with hipMalloc with error %s\n", hipGetErrorString(cerr));
    }

    memset(h_a, 0, bytes);
    cerr = hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    if (cerr != hipSuccess) {
        fprintf(stderr, "Error with Host to Device hipMemcpy with error %s\n", hipGetErrorString(cerr));
    }
    cerr = hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    if (cerr != hipSuccess) {
        fprintf(stderr, "Error with Device to Host hipMemcpy with error %s\n", hipGetErrorString(cerr));
    }
    cerr = hipFree(d_a);
    if (cerr != hipSuccess) {
        fprintf(stderr, "Error with hipFree with error %s\n", hipGetErrorString(cerr));
    }
    cerr = hipProfilerStop();
    if (cerr != hipSuccess) {
        fprintf(stderr, "Error with cudaProfileStop with error %s\n", hipGetErrorString(cerr));
    }
    return 0;
}
